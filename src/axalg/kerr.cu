
#include <hip/hip_runtime.h>
__global__ void gpu_kerr(float *buffer) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 1024) {
        buffer[idx] = buffer[idx] * 2.0f;
    }
}